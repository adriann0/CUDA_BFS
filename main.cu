
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>

#include "testing.cuh"

void printUsage(char *filename)
{
	std::cout << filename << " [filename] [start vertex number] [cpu tests count] [gpu tests count]" << std::endl;

	std::cout << "CSR format. File should contain in the first line number of vertices. Second line is offset row and third neighbours" << std::endl;
}

int main(int argc, char **argv)
{
	if (argc != 5) {
		printUsage(argv[0]);
		return -1;
	}

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cout << "hipSetDevice failed";
		return 1;
	}

	char *filename = argv[1];
	unsigned int startVetex = atoi(argv[2]);
	unsigned int cpuTests = atoi(argv[3]);
	unsigned int gpuTests = atoi(argv[4]);

	std::ifstream file;
	file.open(filename);

	if (!file.is_open())
	{
		std::cerr << "File not opened" << std::endl;
		return -1;
	}

	unsigned int nodes;
	file >> nodes;

	unsigned int *offsets = new unsigned int[nodes + 1];

	for (unsigned int i = 0; i < nodes + 1; i++)
	{
		file >> offsets[i];
	}

	unsigned int *neigbours = new unsigned int[offsets[nodes]];

	for (unsigned int i = 0; i < offsets[nodes]; i++)
	{
		file >> neigbours[i];
	}

	runTests(offsets, neigbours, nodes, startVetex, cpuTests, gpuTests);

	delete[] offsets;
	delete[] neigbours;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
		std::cout << "hipDeviceReset failed!";
        return 1;
    }

    return 0;
}